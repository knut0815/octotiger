
#include <hip/hip_runtime.h>
#ifdef OCTOTIGER_CUDA_ENABLED
#include "cuda_kernel_methods.hpp"
namespace octotiger {
namespace fmm {
    namespace multipole_interactions {
        __global__ void cuda_multipole_interactions_kernel(double* center_of_masses,
            double* multipoles, double* potential_expansions, double* angular_corrections,
            octotiger::fmm::multiindex<> stencil, bool* stencil_phases,
            double* factor_half, double* factor_sixth) {
            octotiger::fmm::multiindex<> cell_index_unpadded(threadIdx.x, threadIdx.y, threadIdx.z);
            printf("yay");
        }
    }    // namespace multipole_interactions
}    // namespace fmm
}    // namespace octotiger
#endif